#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"



double c_x_min;
double c_x_max;
double c_y_min;
double c_y_max;

double pixel_width;
double pixel_height;

int iteration_max = 200;

int image_size;
unsigned char *image_buffer_host;
unsigned char *image_buffer_device;

int i_x_max;
int i_y_max;
int image_buffer_size;

int rgb_size = 3;

int gradient_size = 16;
int colors[17][3] = {
                        {66, 30, 15},
                        {25, 7, 26},
                        {9, 1, 47},
                        {4, 4, 73},
                        {0, 7, 100},
                        {12, 44, 138},
                        {24, 82, 177},
                        {57, 125, 209},
                        {134, 181, 229},
                        {211, 236, 248},
                        {241, 233, 191},
                        {248, 201, 95},
                        {255, 170, 0},
                        {204, 128, 0},
                        {153, 87, 0},
                        {106, 52, 3},
                        {16, 16, 16},
                    };



void allocate_image_buffer(){
    image_buffer_host = (unsigned char *) malloc(sizeof(unsigned char) * image_buffer_size * rgb_size);

    // for(int i = 0; i < image_buffer_size; i++){
    //     image_buffer[i] = (unsigned char *) malloc(sizeof(unsigned char) * rgb_size);
    // };
};

void init(int argc, char *argv[]){
    if(argc < 6){
        printf("usage: ./mandelbrot_seq c_x_min c_x_max c_y_min c_y_max image_size\n");
        printf("examples with image_size = 11500:\n");
        printf("    Full Picture:         ./mandelbrot_seq -2.5 1.5 -2.0 2.0 11500\n");
        printf("    Seahorse Valley:      ./mandelbrot_seq -0.8 -0.7 0.05 0.15 11500\n");
        printf("    Elephant Valley:      ./mandelbrot_seq 0.175 0.375 -0.1 0.1 11500\n");
        printf("    Triple Spiral Valley: ./mandelbrot_seq -0.188 -0.012 0.554 0.754 11500\n");
        exit(0);
    }
    else{
        sscanf(argv[1], "%lf", &c_x_min);
        sscanf(argv[2], "%lf", &c_x_max);
        sscanf(argv[3], "%lf", &c_y_min);
        sscanf(argv[4], "%lf", &c_y_max);
        sscanf(argv[5], "%d", &image_size);

        i_x_max           = image_size;
        i_y_max           = image_size;
        image_buffer_size = image_size * image_size;

        pixel_width       = (c_x_max - c_x_min) / i_x_max;
        pixel_height      = (c_y_max - c_y_min) / i_y_max;
    };
};

// void update_rgb_buffer(int iteration, int x, int y){
//     int color;

//     if(iteration == iteration_max){
//         image_buffer[(i_y_max * y) + x][0] = colors[gradient_size][0];
//         image_buffer[(i_y_max * y) + x][1] = colors[gradient_size][1];
//         image_buffer[(i_y_max * y) + x][2] = colors[gradient_size][2];
//     }
//     else{
//         color = iteration % gradient_size;

//         image_buffer[(i_y_max * y) + x][0] = colors[color][0];
//         image_buffer[(i_y_max * y) + x][1] = colors[color][1];
//         image_buffer[(i_y_max * y) + x][2] = colors[color][2];
//     };
// };

void write_to_file(){
    FILE * file;
    const char * filename               = "output.ppm";
    const char * comment                = "# ";

    int max_color_component_value = 255;

    file = fopen(filename,"wb");

    fprintf(file, "P6\n %s\n %d\n %d\n %d\n", comment,
            i_x_max, i_y_max, max_color_component_value);

    for(int i = 0; i < image_buffer_size * rgb_size; i++){
        fwrite(image_buffer_host + i, 1, 1, file);
    };

    fclose(file);
};


__global__ void compute_mandelbrot_gpu(double pixel_height, double pixel_width, double c_x_min, double c_y_min, \
                                       int image_size, int iteration_max, unsigned char* image_buffer_device){
    int i_x = threadIdx.x + blockDim.x * blockIdx.x;
    int i_y = threadIdx.y + blockDim.y * blockIdx.y;
    
    // printf("i_x=%d | i_y=%d", i_x, i_y);

    // declaração variáveis para a função update_rgb_buffer
    int color;
    int rgb_size = 3;
    int gradient_size = 16;
    int colors[17][3] = {
                        {66, 30, 15},
                        {25, 7, 26},
                        {9, 1, 47},
                        {4, 4, 73},
                        {0, 7, 100},
                        {12, 44, 138},
                        {24, 82, 177},
                        {57, 125, 209},
                        {134, 181, 229},
                        {211, 236, 248},
                        {241, 233, 191},
                        {248, 201, 95},
                        {255, 170, 0},
                        {204, 128, 0},
                        {153, 87, 0},
                        {106, 52, 3},
                        {16, 16, 16},
                    };
    
    double z_x;
    double z_y;
    double z_x_squared;
    double z_y_squared;
    double escape_radius_squared = 4;

    int iteration;
    // int i_x;

    double c_x;
    double c_y;

    // int i_x_max = image_size;
    int i_y_max = image_size;

    if(i_x < image_size && i_y < image_size){

        c_y = c_y_min + i_y * pixel_height;

        if(fabs(c_y) < pixel_height / 2){
            c_y = 0.0;
        };
        
            c_x         = c_x_min + i_x * pixel_width;

            z_x         = 0.0;
            z_y         = 0.0;

            z_x_squared = 0.0;
            z_y_squared = 0.0;

            for(iteration = 0;
                iteration < iteration_max && \
                ((z_x_squared + z_y_squared) < escape_radius_squared);
                iteration++){
                z_y         = 2 * z_x * z_y + c_y;
                z_x         = z_x_squared - z_y_squared + c_x;

                z_x_squared = z_x * z_x;
                z_y_squared = z_y * z_y;
            };

            
            if(iteration == iteration_max){
                image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 0] = colors[gradient_size][0];
                image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 1] = colors[gradient_size][1];
                image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 2] = colors[gradient_size][2];
            }
            else{
                color = iteration % gradient_size;

                image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 0] = colors[color][0];
                image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 1] = colors[color][1];
                image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 2] = colors[color][2];
            };

            // printf("color= %d | i_y= %d | i_x= %d | i_y_max= %d | iteration= %d | image_buffer_device(0,1,2)= (%u, %u, %u)\n", color, i_y, i_x, i_y_max, iteration, \
            //         image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 0], image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 1], \
            //         image_buffer_device[((i_y_max * i_y) + i_x) * rgb_size + 2]);
    };
}


int main(int argc, char *argv[]){
    init(argc, argv);
    
    allocate_image_buffer();
       
    int dimBlock, dimGrid;
    
    // define estrategia do grid e block: quanto mais proximo de 32 a dimensao do block melhor
    // devido ao warp size
    if(image_size > 32){
        dimBlock = 32;
        dimGrid = (int) (image_size / dimBlock) + 1;
    }
    else{
        dimBlock = image_size;
        dimGrid = 1;
    };

    // printf("dimBlock = %d | dimGrid = %d\n", dimBlock, dimGrid);

    // alocando espaço no device
    hipMalloc((void **)&image_buffer_device, sizeof(unsigned char) * image_buffer_size * rgb_size);
    
    // transferir dados do device para o host com cudaMemcpy
    hipMemcpy(image_buffer_device, image_buffer_host, sizeof(unsigned char) * image_buffer_size * rgb_size, hipMemcpyHostToDevice);
    
    // dimensionamento do grid e do block
    dim3 block(dimBlock, dimBlock);
    dim3 grid(dimGrid, dimGrid);

    // chama função para executar no device (GPU)
    compute_mandelbrot_gpu<<<grid, block>>>(pixel_height, pixel_width, c_x_min, c_y_min, image_size, iteration_max, image_buffer_device);
    hipDeviceSynchronize();

    // passando os dados do array image_buffer do device para o host
    hipMemcpy(image_buffer_host, image_buffer_device, sizeof(unsigned char) * image_buffer_size * rgb_size, hipMemcpyDeviceToHost);

    hipFree(image_buffer_device);

    hipDeviceReset();

    write_to_file();

    return 0;
};
